#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <cmath>
#include "SceneContainer.hpp"
#include "Ray.hpp"
#include "Hit.hpp"
#include "ray_defs.hpp"

#include "DistanceRenderer.hpp"

using namespace std;


namespace mm_ray {

  template<typename Accel>
  __global__
  void render_pixel(Scene const& scene,
		    Accel const& objects,
		    int scene_x,
		    int scene_y,
		    Real_t* pixel_out){
  
    /* 
       Renders a single pixel in the image
    */
    int px = threadIdx.x + blockDim.x * blockIdx.x;
    int py = threadIdx.y + blockDim.y * blockIdx.y;
  
    Real_t norm_i = ((px / (Real_t)scene.output[0]) - .5) * scene.viewport[0];
    Real_t norm_j = ((py / (Real_t)scene.output[1]) - .5) * scene.viewport[1];
  
    Vec3 direc;
    direc = norm_i * scene.cam_right + norm_j * scene.cam_up + scene.cam_dir;

    //Normalize ray
    direc = direc / mag(direc);

    Ray ray(direc, scene.cam_loc);

    //Run our ray tracing algorithm

    Hit prop;
    objects.intersect(ray, prop);

    int pix_ix = (py * scene_x + px) * 3;

    //Use this mapping to map 0 to infinity to 0 to 1
    Real_t distance = 1 / (1 + prop.distance);
    pixel_out[pix_ix] = distance;
    pixel_out[pix_ix + 1] = distance;
    pixel_out[pix_ix + 2] = distance;

  }


  template<typename Accelerator>
  void DistanceRenderer<Accelerator>::Render(){
  
    int image_size_x = this->host_scene.output[0];
    int image_size_y = this->host_scene.output[1];


    //A block is a 16 x 16 chunk
    dim3 block;
    //A grid is a bunch of blocks in a chunk of our image
    dim3 grid(image_size_x, image_size_y);

    Real_t* device_pixel_buffer;
    hipMalloc(&device_pixel_buffer, image_size_x * image_size_y * 3 * sizeof(Real_t));

    //Finally run the raytracing kernel
    render_pixel<Accelerator><<<grid, block>>>(this->host_scene,
					       this->host_accel,
					       image_size_x,
					       image_size_y,
					       device_pixel_buffer);

    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
      {
	// print the CUDA error message
	printf("CUDA error: %s\n", hipGetErrorString(error));
      }

  
    //Copy the entire buffer to a temporary buffer
    int buffer_size = image_size_x * image_size_y * 3;
    vector<Real_t> tmp_buffer;
    tmp_buffer.resize(buffer_size);
  
    hipMemcpy(&*(tmp_buffer.begin()), device_pixel_buffer,
	       sizeof(Real_t) * buffer_size,
	       hipMemcpyDeviceToHost);

  
    this->output_buffer.resize(this->host_scene.output[1] * this->host_scene.output[0] * 3);
  
    for (int i = 0; i < this->host_scene.output[1]; i++){
      for (int j = 0; j < this->host_scene.output[0] * 3; j++){
	this->output_buffer[i * this->host_scene.output[0] * 3 + j] = tmp_buffer[i * image_size_x * 3 + j];
      }
    }
  
    hipFree(device_pixel_buffer);
  }


  template<typename Accel>
  Renderer<Accel>*
  DistanceBuilder<Accel>::operator()(rapidjson::Value& val_obj, 
				     Scene const& scn,
				     Accel const& accelerator,
				     std::vector<Geometry*>& geometry) const {
    return new DistanceRenderer<Accel>(scn, accelerator);
  }

  template class DistanceBuilder<SceneContainer>;
  template class DistanceRenderer<SceneContainer>;
}

